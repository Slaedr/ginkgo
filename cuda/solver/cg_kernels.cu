#include "hip/hip_runtime.h"
/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2020, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/solver/cg_kernels.hpp"


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>


#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/thread_ids.cuh"


namespace gko {
namespace kernels {
namespace cuda {
/**
 * @brief The CG solver namespace.
 *
 * @ingroup cg
 */
namespace cg {


namespace syn {
template <typename ValueType>
struct scalar_t {
    matrix::Dense<ValueType> *data;
};

template <typename ValueType>
struct vector_t {
    matrix::Dense<ValueType> *data;
};

template <typename ValueType>
struct const_scalar_t {
    const matrix::Dense<ValueType> *data;
};

template <typename ValueType>
struct const_vector_t {
    const matrix::Dense<ValueType> *data;
};

template <typename ValueType>
scalar_t<ValueType> scalar(matrix::Dense<ValueType> *mtx)
{
    return {mtx};
}

template <typename ValueType>
const_scalar_t<ValueType> scalar(const matrix::Dense<ValueType> *mtx)
{
    return {mtx};
}

template <typename ValueType>
vector_t<ValueType> vector(matrix::Dense<ValueType> *mtx)
{
    return {mtx};
}

template <typename ValueType>
const_vector_t<ValueType> vector(const matrix::Dense<ValueType> *mtx)
{
    return {mtx};
}

template <typename ValueType>
struct device_dense {
    cuda_type<ValueType> *__restrict__ data;
    size_type stride;
};

template <typename ValueType>
device_dense<cuda_type<ValueType>> map_to_device(vector_t<ValueType> mtx)
{
    return {as_cuda_type(mtx.data->get_values()), mtx.data->get_stride()};
}

template <typename ValueType>
device_dense<cuda_type<const ValueType>> map_to_device(
    const_vector_t<ValueType> mtx)
{
    return {as_cuda_type(mtx.data->get_const_values()), mtx.data->get_stride()};
}

template <typename ValueType>
struct device_scalar {
    ValueType *__restrict__ data;
};

template <typename ValueType>
device_scalar<cuda_type<ValueType>> map_to_device(scalar_t<ValueType> mtx)
{
    return {as_cuda_type(mtx.data->get_values())};
}

template <typename ValueType>
device_scalar<cuda_type<const ValueType>> map_to_device(
    const_scalar_t<ValueType> mtx)
{
    return {as_cuda_type(mtx.data->get_const_values())};
}

stopping_status *map_to_device(Array<stopping_status> &status)
{
    return status.get_data();
}

template <typename ValueType>
__device__ ValueType &unpack_on_device(size_type row, size_type col,
                                       device_dense<ValueType> mtx)
{
    return mtx.data[row * mtx.stride + col];
}

template <typename ValueType>
__device__ ValueType &unpack_on_device(size_type row, size_type col,
                                       device_scalar<ValueType> mtx)
{
    return mtx.data[col];
}

__device__ stopping_status &unpack_on_device(size_type row, size_type col,
                                             stopping_status *status)
{
    return status[col];
}

template <typename Function, typename... Args>
__global__ void generic_2d_kernel(size_type num_rows, size_type num_cols,
                                  Function func, Args... args)
{
    auto col = threadIdx.x + blockDim.x * blockIdx.x;
    auto row = threadIdx.y + blockDim.y * blockIdx.y;
    if (row < num_rows && col < num_cols) {
        func(unpack_on_device(row, col, args)..., row, col);
    }
}

template <typename T>
struct size_extract_helper {};

template <typename ValueType>
struct size_extract_helper<vector_t<ValueType>> {
    static constexpr bool has_size() { return true; }
    static gko::dim<2> get_size(vector_t<ValueType> v)
    {
        return v.data->get_size();
    }
    static bool is_compatible(gko::dim<2> size, vector_t<ValueType> v)
    {
        return get_size(v) == size;
    }
};

template <typename ValueType>
struct size_extract_helper<const_vector_t<ValueType>> {
    static constexpr bool has_size() { return true; }
    static gko::dim<2> get_size(const_vector_t<ValueType> v)
    {
        return v.data->get_size();
    }
    static bool is_compatible(gko::dim<2> size, const_vector_t<ValueType> v)
    {
        return get_size(v) == size;
    }
};

template <typename ValueType>
struct size_extract_helper<scalar_t<ValueType>> {
    static constexpr bool has_size() { return false; }
    static gko::dim<2> get_size(scalar_t<ValueType> v)
    {
        return v.data->get_size();
    }
    static bool is_compatible(gko::dim<2> size, scalar_t<ValueType> v)
    {
        return get_size(v)[1] == size[1];
    }
};

template <typename ValueType>
struct size_extract_helper<const_scalar_t<ValueType>> {
    static constexpr bool has_size() { return false; }
    static gko::dim<2> get_size(const_scalar_t<ValueType> v)
    {
        return v.data->get_size();
    }
    static bool is_compatible(gko::dim<2> size, const_scalar_t<ValueType> v)
    {
        return get_size(v)[1] == size[1];
    }
};

template <>
struct size_extract_helper<Array<stopping_status>> {
    static constexpr bool has_size() { return false; }
    static gko::dim<2> get_size(Array<stopping_status> &status)
    {
        return {1, status.get_num_elems()};
    }
    static bool is_compatible(gko::dim<2> size, Array<stopping_status> &v)
    {
        return get_size(v)[1] == size[1];
    }
};

bool all() { return true; }

template <typename Arg, typename... Args>
bool all(Arg arg, Args... args)
{
    return arg && all(args...);
}

gko::dim<2> find_first_size() { return gko::dim<2>{}; }

template <typename Arg, typename... Args>
gko::dim<2> find_first_size(Arg arg, Args... args)
{
    if (size_extract_helper<Arg>::has_size()) {
        return size_extract_helper<Arg>::get_size(arg);
    } else {
        return find_first_size(args...);
    }
}

template <typename Function, typename... Args>
void dispatch(Function func, Args... args)
{
    auto size = find_first_size(args...);
    constexpr auto x_blocksize = 32;
    constexpr auto y_blocksize = 32;
    GKO_ASSERT(all(size_extract_helper<Args>::is_compatible(size, args)...));
    auto x_blocks = ceildiv(size[1], x_blocksize);
    auto y_blocks = ceildiv(size[0], y_blocksize);
    auto blocks = dim3(x_blocks, y_blocks);
    auto threads = dim3(x_blocksize, y_blocksize);
    generic_2d_kernel<<<blocks, threads>>>(size[0], size[1], func,
                                           map_to_device(args)...);
}

}  // namespace syn


template <typename ValueType>
void initialize(std::shared_ptr<const CudaExecutor> exec,
                const matrix::Dense<ValueType> *b, matrix::Dense<ValueType> *r,
                matrix::Dense<ValueType> *z, matrix::Dense<ValueType> *p,
                matrix::Dense<ValueType> *q, matrix::Dense<ValueType> *prev_rho,
                matrix::Dense<ValueType> *rho,
                Array<stopping_status> *stop_status)
{
    using syn::scalar;
    using syn::vector;
    syn::dispatch(
        [] __device__(auto &b, auto &r, auto &z, auto &p, auto &q,
                      auto &prev_rho, auto &rho, auto &stop_status,
                      size_type row, size_type col) {
            if (row == 0) {
                rho = zero();
                prev_rho = one();
                stop_status.reset();
            }
            r = b;
            z = p = q = zero();
        },
        vector(b), vector(r), vector(z), vector(p), vector(q), scalar(prev_rho),
        scalar(rho), *stop_status);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CG_INITIALIZE_KERNEL);


template <typename ValueType>
void step_1(std::shared_ptr<const CudaExecutor> exec,
            matrix::Dense<ValueType> *p, const matrix::Dense<ValueType> *z,
            const matrix::Dense<ValueType> *rho,
            const matrix::Dense<ValueType> *prev_rho,
            const Array<stopping_status> *stop_status)
{
    using syn::scalar;
    using syn::vector;
    syn::dispatch(
        [] __device__(auto &p, auto &z, auto &rho, auto &prev_rho,
                      auto &stop_status, size_type row, size_type col) {
            if (!stop_status.has_stopped()) {
                auto tmp = prev_rho == zero(prev_rho) ? 0 : rho / prev_rho;
                p = z + tmp * p;
            }
        },
        vector(p), vector(z), scalar(rho), scalar(prev_rho), *stop_status);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CG_STEP_1_KERNEL);


template <typename ValueType>
void step_2(std::shared_ptr<const CudaExecutor> exec,
            matrix::Dense<ValueType> *x, matrix::Dense<ValueType> *r,
            const matrix::Dense<ValueType> *p,
            const matrix::Dense<ValueType> *q,
            const matrix::Dense<ValueType> *beta,
            const matrix::Dense<ValueType> *rho,
            const Array<stopping_status> *stop_status)
{
    using syn::scalar;
    using syn::vector;
    syn::dispatch(
        [] __device__(auto &x, auto &r, auto &p, auto &q, auto &beta, auto &rho,
                      auto &stop_status, size_type row, size_type col) {
            if (!stop_status.has_stopped()) {
                auto tmp = beta == zero(beta) ? zero(beta) : rho / beta;
                x += tmp * p;
                r -= tmp * q;
            }
        },
        vector(x), vector(r), vector(p), vector(q), scalar(beta), scalar(rho),
        *stop_status);
}

GKO_INSTANTIATE_FOR_EACH_VALUE_TYPE(GKO_DECLARE_CG_STEP_2_KERNEL);


}  // namespace cg
}  // namespace cuda
}  // namespace kernels
}  // namespace gko
